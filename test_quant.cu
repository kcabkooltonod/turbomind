// #include <cuda_fp16.h>
#include <hip/hip_fp4.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdint.h>
#include <vector>
#include <random> 

__global__ void compute_max_val(const half* x, float* max_val, int group_size, int n_groups) {
    int group_id = blockIdx.x;
    int tid = threadIdx.x;
    int start_idx = group_id * group_size;
    int idx = start_idx + tid;
    // printf("hello");
    extern __shared__ float shared_max[];
    float local_max = 0.0f;

    if (idx < (group_id + 1) * group_size) {
        local_max = fabsf(__half2float(x[idx]));
    }
    // printf("hello");
    shared_max[tid] = local_max;
    __syncthreads();

    // 归约求最大值
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            if (shared_max[tid + s] > shared_max[tid]) {
                shared_max[tid] = shared_max[tid + s];
            }
        }
        __syncthreads();
    }
    if (tid == 0) {
        max_val[group_id] = fmaxf(shared_max[0], 1e-5f);
    }
}

__device__ __hip_fp4_e2m1 lookup_quantize(float x_scaled) {
    const float sorted_candidates[] = {0.0f, 0.5f, 1.0f, 1.5f, 2.0f, 3.0f, 4.0f, 6.0f};
    float abs_x = fabsf(x_scaled);
    float closest = sorted_candidates[0];
    float min_diff = fabsf(abs_x - closest);

    for (int i = 1; i < 8; ++i) {
        float diff = fabsf(abs_x - sorted_candidates[i]);
        if (diff < min_diff) {
            min_diff = diff;
            closest = sorted_candidates[i];
        }
    }
    if (abs_x > sorted_candidates[7]) closest = sorted_candidates[7];
    // closest = 0;
    return __hip_fp4_e2m1(__float2half_rn(closest * copysignf(1.0f, x_scaled)));
}

// 比较查表法与 Intrinsic 法的比特差异
__global__ void compare_fp4_bits(const __half* input, 
                                uint8_t* lookup_result, 
                                uint8_t* intrinsic_result, 
                                int group_size, 
                                const float* max_vals) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // printf("Thread %d: start\n", idx);

    int group_id = idx / group_size;
    // printf("hello");
    // printf("group_id %d", group_id);
    // printf("gridDim.x : %d", gridDim.x);
    // if (group_id >= gridDim.x) return;

    // 计算缩放因子
    float max_val = max_vals[group_id];
    float scale = max_val / 6.0f;

    // 缩放输入值
    float x_scaled = __half2float(input[idx]) / scale;

    // 查表法量化
    __hip_fp4_e2m1 q_lookup = lookup_quantize(x_scaled);
    // Intrinsic 法直接转换
    __hip_fp4_e2m1 q_intrinsic = __hip_fp4_e2m1(__float2half_rn(x_scaled));

    // 将 FP4 值按比特存储到结果中
    // int byte_idx = idx / 2;
    // int shift = (idx % 2) * 4;
    // printf("hello %d ", idx);
    // lookup_result[byte_idx] |= (static_cast<uint8_t>(q_lookup) & 0xF) << shift;
    lookup_result[idx] |= static_cast<uint8_t>(q_lookup.__x);
    // printf("%x\n",q_lookup.__x);
    // intrinsic_result[byte_idx] |= (static_cast<uint8_t>(q_intrinsic) & 0xF) << shift;
    intrinsic_result[idx] |= static_cast<uint8_t>(q_intrinsic.__x);
}


void check_bit_errors(const uint8_t* d_lookup, 
                     const uint8_t* d_intrinsic, 
                     int total_elements) {
    std::vector<uint8_t> h_lookup(total_elements / 2);
    std::vector<uint8_t> h_intrinsic(total_elements / 2);
    hipMemcpy(h_lookup.data(), d_lookup, total_elements / 2, hipMemcpyDeviceToHost);
    hipMemcpy(h_intrinsic.data(), d_intrinsic, total_elements / 2, hipMemcpyDeviceToHost);

    // 打印 h_lookup 和 h_intrinsic 的内容
    printf("\n=== h_lookup (查表法) ===\n");
    for (size_t i = 0; i < h_lookup.size(); ++i) {
        printf("%02X ", h_lookup[i]);
        if ((i + 1) % 8 == 0) printf("\n");  // 每8个字节换行
    }

    printf("\n\n=== h_intrinsic (Intrinsic法) ===\n");
    for (size_t i = 0; i < h_intrinsic.size(); ++i) {
        printf("%02X ", h_intrinsic[i]);
        if ((i + 1) % 8 == 0) printf("\n");
    }
    printf("\n");

    int errors = 0;
    for (size_t i = 0; i < h_lookup.size(); ++i) {
        if (h_lookup[i] != h_intrinsic[i]) {
            // 检查每个 4-bit 单元
            uint8_t lookup_low = h_lookup[i] & 0xF;
            uint8_t intrinsic_low = h_intrinsic[i] & 0xF;
            uint8_t lookup_high = (h_lookup[i] >> 4) & 0xF;
            uint8_t intrinsic_high = (h_intrinsic[i] >> 4) & 0xF;

            if (lookup_low != intrinsic_low) errors++;
            if (lookup_high != intrinsic_high) errors++;
        }
    }
    printf("Bit-level errors: %d/%d (%.2f%%)\n", 
           errors, total_elements, 100.0f * errors / total_elements);
}


int main() {
    const int group_size = 16;
    const int n_groups = 4;
    const int total_elements = group_size * n_groups;

    // 初始化输入数据（正态分布）
    std::random_device rd;
    std::mt19937 gen(rd());
    
    const float mu = 0.0f;      // 均值
    const float sigma = 1.666f; // 标准差

    std::normal_distribution<float> dist(mu, sigma);
    std::vector<__half> h_input(total_elements);

    for (auto& val : h_input) {
        float num = dist(gen);
        val = __float2half_rn(num);
        // val = 100;
    }

    h_input = {
        // 第一行数据
        __half{1.9268f}, __half{1.4873f}, __half{0.90088f}, __half{-2.1055f},
        __half{0.67822f}, __half{-1.2344f}, __half{-0.04306f}, __half{-1.6045f},
        __half{-0.75195f}, __half{1.6484f}, __half{-0.39258f}, __half{-1.4033f},
        __half{-0.72803f}, __half{-0.55957f}, __half{-0.76904f}, __half{0.76221f},
        __half{1.6426f}, __half{-0.15955f}, __half{-0.49731f}, __half{0.4397f},
        __half{-0.7583f}, __half{1.0781f}, __half{0.80078f}, __half{1.6807f},
        __half{1.2793f}, __half{1.2969f}, __half{0.61035f}, __half{1.335f},
        __half{-0.23157f}, __half{0.041748f}, __half{-0.25146f}, __half{0.85986f},
    
        // 第二行数据
        __half{-1.3848f}, __half{-0.87109f}, __half{-0.22339f}, __half{1.7178f},
        __half{0.31885f}, __half{-0.42456f}, __half{0.30566f}, __half{-0.77441f},
        __half{-1.5576f}, __half{0.99561f}, __half{-0.87988f}, __half{-0.60107f},
        __half{-1.2744f}, __half{2.123f}, __half{-1.2344f}, __half{-0.48779f},
        __half{-0.91406f}, __half{-0.6582f}, __half{0.078003f}, __half{0.52588f},
        __half{-0.48804f}, __half{1.1914f}, __half{-0.81396f}, __half{-0.73584f},
        __half{-1.4033f}, __half{0.036011f}, __half{-0.063477f}, __half{0.67578f},
        __half{-0.097778f}, __half{1.8447f}, __half{-1.1846f}, __half{1.3838f},

    };

    __half* d_input;
    float* d_max_vals;
    uint8_t* d_lookup_result, *d_intrinsic_result;

    hipMalloc(&d_input, total_elements * sizeof(__half));
    hipMalloc(&d_max_vals, n_groups * sizeof(float));
    hipMalloc(&d_lookup_result, total_elements);
    hipMalloc(&d_intrinsic_result, total_elements);
    
    hipMemset(d_lookup_result, 0, total_elements);
    hipMemset(d_intrinsic_result, 0, total_elements);

    hipMemcpy(d_input, h_input.data(), total_elements * sizeof(__half), hipMemcpyHostToDevice);

    dim3 block(256);
    dim3 grid((total_elements + block.x - 1) / block.x);

    // 计算group-wise的最大值
    compute_max_val<<<n_groups, group_size, group_size * sizeof(float)>>>(d_input, d_max_vals, group_size, n_groups);
    hipDeviceSynchronize();
    hipError_t err = hipPeekAtLastError();
    if (err != hipSuccess) {
        printf("Kernel launch error: %s\n", hipGetErrorString(err));
    }
    // printf("hello");
    //分别计算查表法和intrinsic法的结果
    compare_fp4_bits<<<grid, block>>>(d_input, d_lookup_result, d_intrinsic_result, group_size, d_max_vals);
    hipDeviceSynchronize();
    err = hipPeekAtLastError();
    if (err != hipSuccess) {
        printf("Kernel launch error: %s\n", hipGetErrorString(err));
    }
    //比较两者的bit-level error
    check_bit_errors(d_lookup_result, d_intrinsic_result, total_elements);
    // Bit-level errors: 0/256 (0.00%)
    hipFree(d_input);
    hipFree(d_max_vals);
    hipFree(d_lookup_result);
    hipFree(d_intrinsic_result);

    return 0;
}

// === h_lookup (查表法) ===
// 07 06 05 0F 04 0E 08 0E 
// 0C 06 0A 0E 0C 0B 0C 04 
// 07 09 0C 03 0D 06 05 07 
// 06 06 04 06 0A 00 0A 05 


// === h_intrinsic (Intrinsic法) ===
// 07 06 05 0F 04 0E 08 0E 
// 0C 06 0A 0E 0C 0B 0C 04 
// 07 09 0C 03 0D 06 05 07 
// 06 06 04 06 0A 00 0A 05 

// Bit-level errors: 0/64 (0.00%)